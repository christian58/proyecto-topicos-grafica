#include "hip/hip_runtime.h"
//nvcc file.cu -o m `pkg-config opencv --cflags --libs`
//./m colosseo.raw out.raw 1000 1000 5 5 10
#include <opencv2/highgui/highgui.hpp>
#include <bits/stdc++.h>
using namespace cv;
using namespace std;

#define BLOCK_SIZE 16
#define GRID_SIZE 256

//Useful to read Error from CUDA Calls
#define CUDA_CALL(x) {if((x) != hipSuccess){ \
  printf("CUDA error at %s:%d\n",__FILE__,__LINE__); \
  printf("  %s\n", hipGetErrorString(hipGetLastError())); \
  exit(EXIT_FAILURE);}}

// nCentroides
__constant__ int dev_nCentroids;
__constant__ int dev_size;

int PALETTE_BYTES = 0; // nCentroids * sizeof(int)
int IMAGE_BYTES = 0;  // width * height * sizeof(int)

//**********************************
//R,G,B Centroid's triple on device
// nCentroids on GPU is HARDCODED remind to update it manually!
__constant__ int dev_RedCentroid[20];
__constant__ int dev_GreenCentroid[20];
__constant__ int dev_BlueCentroid[20];
//**********************************

//get RGB values from Initial Centroids (RAW format)
bool loadPalette(char* filename, int nCentroids, int* redCentroid, int* greenCentroid, int*  blueCentroid) {
	FILE *imageFile;
	int length = 0;
	imageFile = fopen(filename,"r");
	if (imageFile == NULL) {
		return false;
	} else {
		for (int i = 0; i < nCentroids; i++) {
			// R,G,B Centroid triple, nCentroids long
			redCentroid[i] = fgetc(imageFile);
			greenCentroid[i] = fgetc(imageFile);
			blueCentroid[i] = fgetc(imageFile);
			printf("%d, %d, %d\n",redCentroid[i], greenCentroid[i], blueCentroid[i] );
			length++;
		}
		fclose(imageFile);
		printf("\n");
		//printf("Palette Length: %d\n", length);
		return true;
	}
}

//opencv________________________________
void loadRawImage(char* filename, int* r, int* g, int* b, int size){
	Mat image = imread("img.jpg", CV_LOAD_IMAGE_COLOR);
    imshow( "original", image ); 
    //load IMG
    int cols = image.cols;
    int rows = image.rows;
    cout<<"cols: "<<cols<<endl;
    cout<<"rows: "<<rows<<endl;
    for( int i = 0; i < image.rows; i++ ){
        for( int j = 0; j < image.cols; j++ ){
			int rr = image.at<Vec3b>(i,j)[0];
			int gg = image.at<Vec3b>(i,j)[1];
			int bb = image.at<Vec3b>(i,j)[2];
			r[cols*i+j] = rr;
			g[cols*i+j] = gg;
			b[cols*i+j] = bb;
        }
    }
}

void writeRawImage(char* filename, int* labelArray, int* redCentroid, int* greenCentroid, int* blueCentroid, int size){
	Mat image = imread("img.jpg", CV_LOAD_IMAGE_COLOR);
	Mat new_image = Mat::zeros( image.size(), image.type() );
	int cols = image.cols;
    //int rows = image.rows;
    for( int y = 0; y < image.rows; y++ ){
        for( int x = 0; x < image.cols; x++ ){
        	int i = cols*y+x;
            new_image.at<Vec3b>(y,x)[0] = saturate_cast<uchar>( redCentroid[labelArray[i]] );
        	new_image.at<Vec3b>(y,x)[1] = saturate_cast<uchar>( greenCentroid[labelArray[i]] );
        	new_image.at<Vec3b>(y,x)[2] = saturate_cast<uchar>( blueCentroid[labelArray[i]] );
        }
    }
   imshow( "resultado", new_image );
}
//________________________________
//  Clears arrays before each kernel getClusterLabel iteration
__global__ void clearPaletteArrays(int *dev_sumRed,int *dev_sumGreen,int *dev_sumBlue, int* dev_pixelClusterCounter, int* dev_tempRedCentroid, int* dev_tempGreenCentroid, int* dev_tempBlueCentroid ) {
	// 1 block, 16x16 threads
	int threadID = threadIdx.x + threadIdx.y * blockDim.x;
	if(threadID < dev_nCentroids) {
		// nCentroids long
		dev_sumRed[threadID] = 0;
		dev_sumGreen[threadID] = 0;
		dev_sumBlue[threadID] = 0;
		dev_pixelClusterCounter[threadID] = 0;
		dev_tempRedCentroid[threadID] = 0;
		dev_tempGreenCentroid[threadID] = 0;
		dev_tempBlueCentroid[threadID] = 0;
	}
}

//  Clear label array before each kernel getClusterLabel iteration
__global__ void clearLabelArray(int *dev_labelArray){
	int threadID = (threadIdx.x + blockIdx.x * blockDim.x) + (threadIdx.y + blockIdx.y * blockDim.y) * blockDim.x * gridDim.x;
	if(threadID < dev_size) {
		dev_labelArray[threadID] = 0;
	}
}


/*
 * Finds the minimum distance between each triple dev_Red[i] dev_Green[i] dev_Blue[i] and all centroids.
 * Then saves the equivalent centroid label in dev_labelArray.
 * labelArray is   "width*height" long, monodimensional array
 * INPUT : pixel triple arrays dev_Red, dev_Green, dev_Blue. labelArray that will contains the label for each pixel triple
 */
__global__ void getClusterLabel(int *dev_Red,int *dev_Green,int *dev_Blue,int *dev_labelArray) {
	int threadID = (threadIdx.x + blockIdx.x * blockDim.x) + (threadIdx.y + blockIdx.y * blockDim.y) * blockDim.x * gridDim.x;// Global thread index
	float min = 500.0, value;//default min value of distance
	int index = 0;//will be label
	if(threadID < dev_size) {
		// Finding the nearest centroid to current triple identified by threadID thread
		for(int i = 0; i < dev_nCentroids; i++) {
			// Performing Euclidean distance, Saving current value
			value = sqrtf(powf((dev_Red[threadID]-dev_RedCentroid[i]),2.0) + powf((dev_Green[threadID]-dev_GreenCentroid[i]),2.0) + powf((dev_Blue[threadID]-dev_BlueCentroid[i]),2.0));
			if(value < min){
				// saving new nearest centroid
				min = value;
				// Updating his index
				index = i;
			}
		}// end for
		// Writing to global memory the index of the nearest centroid
		// for dev_Red[threadID], dev_Green[threadID], dev_Blue[threadID] pixel triple
		dev_labelArray[threadID] = index;
	}// end if
}// end getClusterLabel


/*
 *  Summing Red, Green, Blue values per cluster
 *  Counting how many pixels there are in each cluster
 *
 */
__global__ void sumCluster(int *dev_Red,int *dev_Green,int *dev_Blue,int *dev_sumRed,int *dev_sumGreen,int *dev_sumBlue,int *dev_labelArray,int *dev_pixelClusterCounter) {
	// Global thread index
	int threadID = (threadIdx.x + blockIdx.x * blockDim.x) + (threadIdx.y + blockIdx.y * blockDim.y) * blockDim.x * gridDim.x;
	if(threadID < dev_size) {
		int currentLabelArray = dev_labelArray[threadID];
		int currentRed = dev_Red[threadID];
		int currentGreen = dev_Green[threadID];
		int currentBlue = dev_Blue[threadID];
		// Writing to global memory needs a serialization. Many threads are writing into the same few locations
		atomicAdd(&dev_sumRed[currentLabelArray], currentRed);
		atomicAdd(&dev_sumGreen[currentLabelArray], currentGreen);
		atomicAdd(&dev_sumBlue[currentLabelArray], currentBlue);
		atomicAdd(&dev_pixelClusterCounter[currentLabelArray], 1);
	}
}

/*
 *  Calculates the new R,G,B values of the centroids dividing the sum of color (for each channel) by the number of pixels in that cluster
 *	New values are stored in global memory since the current R,G,B values of the centroids are in read-only constant memory.
 */
__global__ void newCentroids(int *dev_tempRedCentroid, int *dev_tempGreenCentroid, int *dev_tempBlueCentroid,int* dev_sumRed, int *dev_sumGreen,int *dev_sumBlue, int* dev_pixelClusterCounter) {
	int threadID = threadIdx.x + threadIdx.y * blockDim.x;
	if(threadID < dev_nCentroids) {
		int currentPixelCounter = dev_pixelClusterCounter[threadID];
		int sumRed = dev_sumRed[threadID];
		int sumGreen = dev_sumGreen[threadID];
		int sumBlue = dev_sumBlue[threadID];
		//new RGB Centroids' values written in global memory
		dev_tempRedCentroid[threadID] = (int)(sumRed/currentPixelCounter);
		dev_tempGreenCentroid[threadID] = (int)(sumGreen/currentPixelCounter);
		dev_tempBlueCentroid[threadID] = (int)(sumBlue/currentPixelCounter);
	}
}


int main(int argc, char *argv[]) {

		Mat image = imread("img.jpg", CV_LOAD_IMAGE_COLOR);
    	imshow( "original", image ); 
		// init device
		hipSetDevice(0);
		hipDeviceSynchronize();
		hipDeviceSynchronize();

		//input raw file, output raw file, input palette raw file containing RGB values of initial centroids
		char *inputFile, *outputFile, *palette;
		//Pixels' r,g,b values. Centroid's r,g,b values
		int *red, *green, *blue, *redCentroid, *greenCentroid, *blueCentroid;

		// ref to GPU  Pixels'RGB values, Centroids' RGB values
		int *dev_Red, *dev_Green, *dev_Blue, *dev_tempRedCentroid, *dev_tempGreenCentroid, *dev_tempBlueCentroid;
		// array containing ref to GPU label array variable
		int *labelArray, *dev_labelArray;

		// local variables for storing image width, height
		// number of cluster, number of iterations, linear size of the image ( = width * height)
		int width, height, nCentroids, nIterations,size;
		//int IMAGE_BYTES, PALETTE_BYTES;

		// ref to array where pixels' count are stored
		int *pixelClusterCounter, *dev_pixelClusterCounter;
		// ref to arrays where sum of RGB values for each cluster are stored
		int *sumRed, *sumGreen, *sumBlue;
		int *dev_sumRed, *dev_sumGreen, *dev_sumBlue;

		// checking arguments
		if (argc > 7) {
			inputFile = argv[1];
			outputFile = argv[2];
			width = atoi(argv[3]);
			height = atoi(argv[4]);
			palette = argv[5];
			nCentroids = atoi(argv[6]);  // remind to update hardcoded nCentroids above
			if(nCentroids > 256)
				nCentroids = 256;
			nIterations = atoi(argv[7]);
			if(nIterations > 15)
				nIterations = 15;

		} else {
			printf("  USAGE: kmeans.cu <inputfile.raw> <outputfile.raw> nRows nCols palette nCentroids nItarations \n");
			printf("           <inputfile.raw>: input .raw file (sequence of bytes)\n");
			printf("          <outputfile.raw>: output .raw file\n");
			printf("                  nRows: the number of rows of the image\n");
			printf("                  nCols: the number of columns of the image\n");
			printf("			      palette: RGB initial Centroids");
			printf("			   nCentroids: number of clusters");
			printf("			  nIterations: number of iterations of K-Means");

			return 0;
		}

		// Setting image and palette size in bytes
		IMAGE_BYTES = width * height * sizeof(int);
		PALETTE_BYTES = nCentroids * sizeof(int);
		size = width * height;

		printf("Image: %s\n",inputFile);
		printf("Width: %d, Height: %d\n", width, height);
		printf("#Clusters: %d, #Iterations: %d\n", nCentroids, nIterations);


		// allocate memory on CPU
		red = static_cast<int *>(malloc(IMAGE_BYTES));
		green = static_cast<int *>(malloc(IMAGE_BYTES));
		blue = static_cast<int *>(malloc(IMAGE_BYTES));
		redCentroid = static_cast<int *>(malloc(PALETTE_BYTES));
		greenCentroid = static_cast<int *>(malloc(PALETTE_BYTES));
		blueCentroid = static_cast<int *>(malloc(PALETTE_BYTES));
		labelArray = static_cast<int *>(malloc(IMAGE_BYTES));
		sumRed = static_cast<int*>(malloc(PALETTE_BYTES));
		sumGreen = static_cast<int*>(malloc(PALETTE_BYTES));
		sumBlue = static_cast<int*>(malloc(PALETTE_BYTES));
		pixelClusterCounter = static_cast<int*>(malloc(PALETTE_BYTES));

		//centroides iniciales
		printf("Initial Centroids: \n");
		loadPalette(palette, nCentroids, redCentroid, greenCentroid, blueCentroid);
		// Cargar imagen en arrays r, g, b
		loadRawImage(inputFile, red, green, blue, size);

		if(IMAGE_BYTES == 0 || PALETTE_BYTES == 0) {
			return -1;
		}
		// allocate memory on GPU
		CUDA_CALL(hipMalloc((void**) &dev_Red, IMAGE_BYTES));
		CUDA_CALL(hipMalloc((void**) &dev_Green, IMAGE_BYTES));
		CUDA_CALL(hipMalloc((void**) &dev_Blue, IMAGE_BYTES));
		CUDA_CALL(hipMalloc((void**) &dev_tempRedCentroid, PALETTE_BYTES));
		CUDA_CALL(hipMalloc((void**) &dev_tempGreenCentroid, PALETTE_BYTES));
		CUDA_CALL(hipMalloc((void**) &dev_tempBlueCentroid, PALETTE_BYTES));
		CUDA_CALL(hipMalloc((void**) &dev_labelArray, IMAGE_BYTES));
		CUDA_CALL(hipMalloc((void**) &dev_sumRed, PALETTE_BYTES));
		CUDA_CALL(hipMalloc((void**) &dev_sumGreen, PALETTE_BYTES));
		CUDA_CALL(hipMalloc((void**) &dev_sumBlue, PALETTE_BYTES));
		CUDA_CALL(hipMalloc((void**) &dev_pixelClusterCounter, PALETTE_BYTES));
		// copy host CPU memory to GPU
		CUDA_CALL(hipMemcpy(dev_Red, red, IMAGE_BYTES, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dev_Green, green, IMAGE_BYTES, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dev_Blue, blue, IMAGE_BYTES, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dev_tempRedCentroid, redCentroid,PALETTE_BYTES,hipMemcpyHostToDevice ));
		CUDA_CALL(hipMemcpy(dev_tempGreenCentroid, greenCentroid,PALETTE_BYTES,hipMemcpyHostToDevice ));
		CUDA_CALL(hipMemcpy(dev_tempBlueCentroid, blueCentroid,PALETTE_BYTES,hipMemcpyHostToDevice ));
		CUDA_CALL(hipMemcpy(dev_labelArray, labelArray, IMAGE_BYTES, hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(dev_sumRed, sumRed, PALETTE_BYTES, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dev_sumGreen, sumGreen, PALETTE_BYTES, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dev_sumBlue, sumBlue, PALETTE_BYTES, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dev_pixelClusterCounter, pixelClusterCounter, PALETTE_BYTES, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_RedCentroid), redCentroid, PALETTE_BYTES));
		CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_GreenCentroid), greenCentroid, PALETTE_BYTES));
		CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_BlueCentroid), blueCentroid, PALETTE_BYTES));
		CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_nCentroids),&nCentroids, sizeof(int)));
		CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_size), &size, sizeof(int)));

		// Clearing centroids on host
		for(int i = 0; i < nCentroids; i++) {
			redCentroid[i] = 0;
			greenCentroid[i] = 0;
			blueCentroid[i] = 0;
		}

		// Defining grid size
		int BLOCK_X, BLOCK_Y;
		BLOCK_X = ceil(width/BLOCK_SIZE);
		BLOCK_Y = ceil(height/BLOCK_SIZE);
		if(BLOCK_X > GRID_SIZE)
			BLOCK_X = GRID_SIZE;
		if(BLOCK_Y > GRID_SIZE)
			BLOCK_Y = GRID_SIZE;

		//2D Grid
		//Minimum number of threads that can handle width�height pixels
	 	dim3 dimGRID(BLOCK_X,BLOCK_Y);
	 	//2D Block
	 	//Each dimension is fixed
		dim3 dimBLOCK(BLOCK_SIZE,BLOCK_SIZE);

		printf("Launching K-Means Kernels..	\n");
		//Iteration of kmeans algorithm
		for(int i = 0; i < nIterations; i++) {
			cout<<"Iteracion: "<<i<<endl;
			// Passing image RGB components, palette RGB components, label Array, number of Clusters
			// Init  arrays' values to 0
			// Kernel needs only 1 block since nClusters
			clearPaletteArrays<<<1, dimBLOCK>>>(dev_sumRed, dev_sumGreen, dev_sumBlue, dev_pixelClusterCounter, dev_tempRedCentroid, dev_tempGreenCentroid, dev_tempBlueCentroid);

			// Init labelarray values to 0
			clearLabelArray<<<dimGRID, dimBLOCK>>>(dev_labelArray);

			// Calculates the distance from each pixel and all centroids
			// Then saves the equivalent label in dev_labelArray
			getClusterLabel<<< dimGRID, dimBLOCK >>> (dev_Red, dev_Green, dev_Blue,dev_labelArray);

			//Sums RGB values in each Cluster
			sumCluster<<<dimGRID, dimBLOCK>>> (dev_Red, dev_Green, dev_Blue, dev_sumRed, dev_sumGreen, dev_sumBlue, dev_labelArray,dev_pixelClusterCounter);

			//Finds new RGB Centroids' values
			newCentroids<<<1,dimBLOCK >>>(dev_tempRedCentroid, dev_tempGreenCentroid, dev_tempBlueCentroid, dev_sumRed, dev_sumGreen, dev_sumBlue, dev_pixelClusterCounter);

			//Old RGB Centroids' values are in constant memory
			//Updated RGB Centroids' values are in global memory
			//We need a swap
			CUDA_CALL(hipMemcpy(redCentroid, dev_tempRedCentroid, PALETTE_BYTES,hipMemcpyDeviceToHost));
			CUDA_CALL(hipMemcpy(greenCentroid, dev_tempGreenCentroid, PALETTE_BYTES,hipMemcpyDeviceToHost));
			CUDA_CALL(hipMemcpy(blueCentroid, dev_tempBlueCentroid, PALETTE_BYTES,hipMemcpyDeviceToHost));
			//Uploading in constant memory updated RGB Centroids' values
			CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_RedCentroid), redCentroid, PALETTE_BYTES));
			CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_GreenCentroid), greenCentroid, PALETTE_BYTES));
			CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_BlueCentroid), blueCentroid, PALETTE_BYTES));
		}

		// DEBUG
		CUDA_CALL(hipMemcpy(labelArray, dev_labelArray, IMAGE_BYTES, hipMemcpyDeviceToHost));
		CUDA_CALL(hipMemcpy(sumRed, dev_sumRed, PALETTE_BYTES, hipMemcpyDeviceToHost));
		CUDA_CALL(hipMemcpy(sumGreen, dev_sumGreen, PALETTE_BYTES, hipMemcpyDeviceToHost));
		CUDA_CALL(hipMemcpy(sumBlue, dev_sumBlue, PALETTE_BYTES, hipMemcpyDeviceToHost));
		CUDA_CALL(hipMemcpy(pixelClusterCounter, dev_pixelClusterCounter, PALETTE_BYTES, hipMemcpyDeviceToHost));

		//printf("Kmeans code ran in: %f msecs.\n", timer.Elapsed());
		printf("\n");

		// labelArray DEBUG
		int counter = 0;

		printf("Label Array:\n");
		for(int i = 0; i < (size); i++) {
			//printf("%d\n", labelArray[i]);
			counter++;
		}
		printf("printing counter %d\n", counter);
		counter = 0;

		printf("Sum Arrays:\n");
		for(int j = 0; j < nCentroids; j++) {
			printf("r: %u g: %u b: %u \n", sumRed[j], sumGreen[j], sumBlue[j]);
			counter++;
		}

		printf("\n");

		printf("Pixels per centroids:\n");
		for(int k = 0; k < nCentroids; k++){
			printf("%d centroid: %d pixels\n", k, pixelClusterCounter[k]);
		}
		printf("\n");
		printf("New centroids:\n");
		for(int i = 0; i < nCentroids; i++) {
			printf("%d, %d, %d \n", redCentroid[i], greenCentroid[i], blueCentroid[i]);
		}
		//escribiendo
		cout<<"guardando Imagen"<<endl;
		writeRawImage(outputFile,labelArray, redCentroid, greenCentroid,  blueCentroid,  size);

		free(red);
		free(green);
		free(blue);
		free(redCentroid);
		free(greenCentroid);
		free(blueCentroid);
		free(labelArray);
		free(sumRed);
		free(sumGreen);
		free(sumBlue);
		free(pixelClusterCounter);

		CUDA_CALL(hipFree(dev_Red));
		CUDA_CALL(hipFree(dev_Green));
		CUDA_CALL(hipFree(dev_Blue));
		CUDA_CALL(hipFree(dev_tempRedCentroid));
		CUDA_CALL(hipFree(dev_tempGreenCentroid));
		CUDA_CALL(hipFree(dev_tempBlueCentroid));
		CUDA_CALL(hipFree(dev_labelArray));
		CUDA_CALL(hipFree(dev_sumRed));
		CUDA_CALL(hipFree(dev_sumGreen));
		CUDA_CALL(hipFree(dev_sumBlue));
		CUDA_CALL(hipFree(dev_pixelClusterCounter));

		printf("That's the end.\n");

	
	//waitKey(0);
	double fps = 60;
    // calcular el tiempo de espera entre cada imagen a mostrar
    int delay = 1000 / fps;
    while (true){
        if(waitKey(delay) == 27) break;
    }
	return 0;
}


