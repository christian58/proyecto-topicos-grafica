#include "hip/hip_runtime.h"
//nvcc seg.cu -o m `pkg-config opencv --cflags --libs`
//./m
#include <opencv2/highgui/highgui.hpp>
#include <bits/stdc++.h>
using namespace cv;
using namespace std;

#define BLOCK_SIZE 16
#define GRID_SIZE 256

//Useful to read Error from CUDA Calls
#define CUDA_CALL(x) {if((x) != hipSuccess){ \
  printf("CUDA error at %s:%d\n",__FILE__,__LINE__); \
  printf("  %s\n", hipGetErrorString(hipGetLastError())); \
  exit(EXIT_FAILURE);}}

// nCentroides
__constant__ int dev_nCentroids;
__constant__ int dev_size;

int PALETTE_BYTES = 0; // nCentroids * sizeof(int)
int IMAGE_BYTES = 0;  // width * height * sizeof(int)

//**********************************
//R,G,B Centroid's triple on device
// nCentroids on GPU is HARDCODED remind to update it manually!
__constant__ int dev_RedCentroid[20];
__constant__ int dev_GreenCentroid[20];
__constant__ int dev_BlueCentroid[20];
//**********************************
//opencv________________________________
void loadRawImage(string filename, int* r, int* g, int* b, int size){
	Mat image = imread(filename, CV_LOAD_IMAGE_COLOR);
    imshow( "original", image ); 
    //load IMG
    int cols = image.cols;
    int rows = image.rows;
    cout<<"cols: "<<cols<<endl;
    cout<<"rows: "<<rows<<endl;
    for( int i = 0; i < image.rows; i++ ){
        for( int j = 0; j < image.cols; j++ ){
			int rr = image.at<Vec3b>(i,j)[0];
			int gg = image.at<Vec3b>(i,j)[1];
			int bb = image.at<Vec3b>(i,j)[2];
			r[cols*i+j] = rr;
			g[cols*i+j] = gg;
			b[cols*i+j] = bb;
        }
    }
}

//mostrar y guardar imagen
void imprimir_resultado(string filename, int* labelArray, int* redCentroid, int* greenCentroid, int* blueCentroid, int size){
	Mat image = imread(filename, CV_LOAD_IMAGE_COLOR);
	Mat new_image = Mat::zeros( image.size(), image.type() );
	int cols = image.cols;
    //int rows = image.rows;
    for( int y = 0; y < image.rows; y++ ){
        for( int x = 0; x < image.cols; x++ ){
        	int i = cols*y+x;
            new_image.at<Vec3b>(y,x)[0] = saturate_cast<uchar>( redCentroid[labelArray[i]] );
        	new_image.at<Vec3b>(y,x)[1] = saturate_cast<uchar>( greenCentroid[labelArray[i]] );
        	new_image.at<Vec3b>(y,x)[2] = saturate_cast<uchar>( blueCentroid[labelArray[i]] );
        }
    }
    imwrite("out_"+filename,new_image);
   	imshow( "resultado", new_image );
}
//________________________________
//  Clears arrays before each kernel getClusterLabel iteration
__global__ void clearPaletteArrays(int *dev_sumRed,int *dev_sumGreen,int *dev_sumBlue, int* dev_pixelClusterCounter, int* dev_tempRedCentroid, int* dev_tempGreenCentroid, int* dev_tempBlueCentroid ) {
	// 1 block, 16x16 threads
	int threadID = threadIdx.x + threadIdx.y * blockDim.x;
	if(threadID < dev_nCentroids) {
		// nCentroids long
		dev_sumRed[threadID] = 0;
		dev_sumGreen[threadID] = 0;
		dev_sumBlue[threadID] = 0;
		dev_pixelClusterCounter[threadID] = 0;
		dev_tempRedCentroid[threadID] = 0;
		dev_tempGreenCentroid[threadID] = 0;
		dev_tempBlueCentroid[threadID] = 0;
	}
}

//  Clear label array before each kernel getClusterLabel iteration
__global__ void clearLabelArray(int *dev_labelArray){
	int threadID = (threadIdx.x + blockIdx.x * blockDim.x) + (threadIdx.y + blockIdx.y * blockDim.y) * blockDim.x * gridDim.x;
	if(threadID < dev_size) {
		dev_labelArray[threadID] = 0;
	}
}


/*
 * Finds the minimum distance between each triple dev_Red[i] dev_Green[i] dev_Blue[i] and all centroids.
 * Then saves the equivalent centroid label in dev_labelArray.
 * labelArray is   "width*height" long, monodimensional array
 * INPUT : pixel triple arrays dev_Red, dev_Green, dev_Blue. labelArray that will contains the label for each pixel triple
 */
__global__ void getClusterLabel(int *dev_Red,int *dev_Green,int *dev_Blue,int *dev_labelArray) {
	int threadID = (threadIdx.x + blockIdx.x * blockDim.x) + (threadIdx.y + blockIdx.y * blockDim.y) * blockDim.x * gridDim.x;// Global thread index
	float min = 500.0, value;//default min value of distance
	int index = 0;//will be label
	if(threadID < dev_size) {
		// Finding the nearest centroid to current triple identified by threadID thread
		for(int i = 0; i < dev_nCentroids; i++) {
			// Performing Euclidean distance, Saving current value
			value = sqrtf(powf((dev_Red[threadID]-dev_RedCentroid[i]),2.0) + powf((dev_Green[threadID]-dev_GreenCentroid[i]),2.0) + powf((dev_Blue[threadID]-dev_BlueCentroid[i]),2.0));
			if(value < min){
				// saving new nearest centroid
				min = value;
				// Updating his index
				index = i;
			}
		}// end for
		// Writing to global memory the index of the nearest centroid
		// for dev_Red[threadID], dev_Green[threadID], dev_Blue[threadID] pixel triple
		dev_labelArray[threadID] = index;
	}// end if
}// end getClusterLabel


/*
 *  Summing Red, Green, Blue values per cluster
 *  Counting how many pixels there are in each cluster
 *
 */
__global__ void sumCluster(int *dev_Red,int *dev_Green,int *dev_Blue,int *dev_sumRed,int *dev_sumGreen,int *dev_sumBlue,int *dev_labelArray,int *dev_pixelClusterCounter) {
	// Global thread index
	int threadID = (threadIdx.x + blockIdx.x * blockDim.x) + (threadIdx.y + blockIdx.y * blockDim.y) * blockDim.x * gridDim.x;
	if(threadID < dev_size) {
		int currentLabelArray = dev_labelArray[threadID];
		int currentRed = dev_Red[threadID];
		int currentGreen = dev_Green[threadID];
		int currentBlue = dev_Blue[threadID];
		// Writing to global memory needs a serialization. Many threads are writing into the same few locations
		atomicAdd(&dev_sumRed[currentLabelArray], currentRed);
		atomicAdd(&dev_sumGreen[currentLabelArray], currentGreen);
		atomicAdd(&dev_sumBlue[currentLabelArray], currentBlue);
		atomicAdd(&dev_pixelClusterCounter[currentLabelArray], 1);
	}
}

/*
 *  Calculates the new R,G,B values of the centroids dividing the sum of color (for each channel) by the number of pixels in that cluster
 *	New values are stored in global memory since the current R,G,B values of the centroids are in read-only constant memory.
 */
__global__ void newCentroids(int *dev_tempRedCentroid, int *dev_tempGreenCentroid, int *dev_tempBlueCentroid,int* dev_sumRed, int *dev_sumGreen,int *dev_sumBlue, int* dev_pixelClusterCounter) {
	int threadID = threadIdx.x + threadIdx.y * blockDim.x;
	if(threadID < dev_nCentroids) {
		int currentPixelCounter = dev_pixelClusterCounter[threadID];
		int sumRed = dev_sumRed[threadID];
		int sumGreen = dev_sumGreen[threadID];
		int sumBlue = dev_sumBlue[threadID];
		//new RGB Centroids' values written in global memory
		dev_tempRedCentroid[threadID] = (int)(sumRed/currentPixelCounter);
		dev_tempGreenCentroid[threadID] = (int)(sumGreen/currentPixelCounter);
		dev_tempBlueCentroid[threadID] = (int)(sumBlue/currentPixelCounter);
	}
}





int run(string img_name, int nCentroids, int nIterations) {
		Mat image = imread(img_name, CV_LOAD_IMAGE_COLOR);
    	imshow( "original", image ); 
		// init device
		hipSetDevice(0);
		hipDeviceSynchronize();
		hipDeviceSynchronize();

		//Pixels' r,g,b values. Centroid's r,g,b values
		int *red, *green, *blue, *redCentroid, *greenCentroid, *blueCentroid;

		// ref to GPU  Pixels'RGB values, Centroids' RGB values
		int *dev_Red, *dev_Green, *dev_Blue, *dev_tempRedCentroid, *dev_tempGreenCentroid, *dev_tempBlueCentroid;
		// array containing ref to GPU label array variable
		int *labelArray, *dev_labelArray;

		// local variables for storing image width, height
		// number of cluster, number of iterations, linear size of the image ( = width * height)
		int width, height,size;
		//int IMAGE_BYTES, PALETTE_BYTES;

		// ref to array where pixels' count are stored
		int *pixelClusterCounter, *dev_pixelClusterCounter;
		// ref to arrays where sum of RGB values for each cluster are stored
		int *sumRed, *sumGreen, *sumBlue;
		int *dev_sumRed, *dev_sumGreen, *dev_sumBlue;

		width = image.cols;
		height = image.rows;
		

		// Setting image and palette size in bytes
		IMAGE_BYTES = width * height * sizeof(int);
		PALETTE_BYTES = nCentroids * sizeof(int);
		size = width * height;

		cout<<"Imagen: "<<img_name<<endl;
		printf("Width: %d, Height: %d\n", width, height);
		cout<<"#Clusters: "<<nCentroids<<endl;
		cout<<"#Iteraciones: "<<nIterations<<endl;


		// allocate memory on CPU
		red = static_cast<int *>(malloc(IMAGE_BYTES));
		green = static_cast<int *>(malloc(IMAGE_BYTES));
		blue = static_cast<int *>(malloc(IMAGE_BYTES));
		redCentroid = static_cast<int *>(malloc(PALETTE_BYTES));
		greenCentroid = static_cast<int *>(malloc(PALETTE_BYTES));
		blueCentroid = static_cast<int *>(malloc(PALETTE_BYTES));
		labelArray = static_cast<int *>(malloc(IMAGE_BYTES));
		sumRed = static_cast<int*>(malloc(PALETTE_BYTES));
		sumGreen = static_cast<int*>(malloc(PALETTE_BYTES));
		sumBlue = static_cast<int*>(malloc(PALETTE_BYTES));
		pixelClusterCounter = static_cast<int*>(malloc(PALETTE_BYTES));

		// Cargar imagen en arrays r, g, b
		loadRawImage(img_name, red, green, blue, size);

		if(IMAGE_BYTES == 0 || PALETTE_BYTES == 0) {
			return -1;
		}
		// allocate memory on GPU
		CUDA_CALL(hipMalloc((void**) &dev_Red, IMAGE_BYTES));
		CUDA_CALL(hipMalloc((void**) &dev_Green, IMAGE_BYTES));
		CUDA_CALL(hipMalloc((void**) &dev_Blue, IMAGE_BYTES));
		CUDA_CALL(hipMalloc((void**) &dev_tempRedCentroid, PALETTE_BYTES));
		CUDA_CALL(hipMalloc((void**) &dev_tempGreenCentroid, PALETTE_BYTES));
		CUDA_CALL(hipMalloc((void**) &dev_tempBlueCentroid, PALETTE_BYTES));
		CUDA_CALL(hipMalloc((void**) &dev_labelArray, IMAGE_BYTES));
		CUDA_CALL(hipMalloc((void**) &dev_sumRed, PALETTE_BYTES));
		CUDA_CALL(hipMalloc((void**) &dev_sumGreen, PALETTE_BYTES));
		CUDA_CALL(hipMalloc((void**) &dev_sumBlue, PALETTE_BYTES));
		CUDA_CALL(hipMalloc((void**) &dev_pixelClusterCounter, PALETTE_BYTES));
		// copy host CPU memory to GPU
		CUDA_CALL(hipMemcpy(dev_Red, red, IMAGE_BYTES, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dev_Green, green, IMAGE_BYTES, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dev_Blue, blue, IMAGE_BYTES, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dev_tempRedCentroid, redCentroid,PALETTE_BYTES,hipMemcpyHostToDevice ));
		CUDA_CALL(hipMemcpy(dev_tempGreenCentroid, greenCentroid,PALETTE_BYTES,hipMemcpyHostToDevice ));
		CUDA_CALL(hipMemcpy(dev_tempBlueCentroid, blueCentroid,PALETTE_BYTES,hipMemcpyHostToDevice ));
		CUDA_CALL(hipMemcpy(dev_labelArray, labelArray, IMAGE_BYTES, hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(dev_sumRed, sumRed, PALETTE_BYTES, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dev_sumGreen, sumGreen, PALETTE_BYTES, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dev_sumBlue, sumBlue, PALETTE_BYTES, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dev_pixelClusterCounter, pixelClusterCounter, PALETTE_BYTES, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_RedCentroid), redCentroid, PALETTE_BYTES));
		CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_GreenCentroid), greenCentroid, PALETTE_BYTES));
		CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_BlueCentroid), blueCentroid, PALETTE_BYTES));
		CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_nCentroids),&nCentroids, sizeof(int)));
		CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_size), &size, sizeof(int)));

		// Clearing centroids on host
		for(int i = 0; i < nCentroids; i++) {
			redCentroid[i] = rand()%255;//0;
			greenCentroid[i] = rand()%255;//0;
			blueCentroid[i] = rand()%255;//0;
		}

		printf("\n");
		printf("Centroides Iniciales:\n");
		for(int i = 0; i < nCentroids; i++) {
			printf("%d) [ %d, %d, %d ]\n",i, redCentroid[i], greenCentroid[i], blueCentroid[i]);
		}
		printf("\n");

		// Defining grid size
		int BLOCK_X, BLOCK_Y;
		BLOCK_X = ceil(width/BLOCK_SIZE);
		BLOCK_Y = ceil(height/BLOCK_SIZE);
		if(BLOCK_X > GRID_SIZE)
			BLOCK_X = GRID_SIZE;
		if(BLOCK_Y > GRID_SIZE)
			BLOCK_Y = GRID_SIZE;

		//2D Grid
		//Minimum number of threads that can handle width�height pixels
	 	dim3 dimGRID(BLOCK_X,BLOCK_Y);
	 	//2D Block
	 	//Each dimension is fixed
		dim3 dimBLOCK(BLOCK_SIZE,BLOCK_SIZE);

		printf("Run K-Means Kernels:\n");
		//Iteration of kmeans algorithm
		for(int i = 0; i < nIterations; i++) {
			cout<<"Iteracion: "<<i<<endl;
			// Passing image RGB components, palette RGB components, label Array, number of Clusters
			// Init  arrays' values to 0
			// Kernel needs only 1 block since nClusters
			clearPaletteArrays<<<1, dimBLOCK>>>(dev_sumRed, dev_sumGreen, dev_sumBlue, dev_pixelClusterCounter, dev_tempRedCentroid, dev_tempGreenCentroid, dev_tempBlueCentroid);

			// Init labelarray values to 0
			clearLabelArray<<<dimGRID, dimBLOCK>>>(dev_labelArray);

			// Calculates the distance from each pixel and all centroids
			// Then saves the equivalent label in dev_labelArray
			getClusterLabel<<< dimGRID, dimBLOCK >>> (dev_Red, dev_Green, dev_Blue,dev_labelArray);

			//Sums RGB values in each Cluster
			sumCluster<<<dimGRID, dimBLOCK>>> (dev_Red, dev_Green, dev_Blue, dev_sumRed, dev_sumGreen, dev_sumBlue, dev_labelArray,dev_pixelClusterCounter);

			//Finds new RGB Centroids' values
			newCentroids<<<1,dimBLOCK >>>(dev_tempRedCentroid, dev_tempGreenCentroid, dev_tempBlueCentroid, dev_sumRed, dev_sumGreen, dev_sumBlue, dev_pixelClusterCounter);

			//Old RGB Centroids' values are in constant memory
			//Updated RGB Centroids' values are in global memory
			//We need a swap
			CUDA_CALL(hipMemcpy(redCentroid, dev_tempRedCentroid, PALETTE_BYTES,hipMemcpyDeviceToHost));
			CUDA_CALL(hipMemcpy(greenCentroid, dev_tempGreenCentroid, PALETTE_BYTES,hipMemcpyDeviceToHost));
			CUDA_CALL(hipMemcpy(blueCentroid, dev_tempBlueCentroid, PALETTE_BYTES,hipMemcpyDeviceToHost));
			//Uploading in constant memory updated RGB Centroids' values
			CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_RedCentroid), redCentroid, PALETTE_BYTES));
			CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_GreenCentroid), greenCentroid, PALETTE_BYTES));
			CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_BlueCentroid), blueCentroid, PALETTE_BYTES));
		}

		// DEBUG
		CUDA_CALL(hipMemcpy(labelArray, dev_labelArray, IMAGE_BYTES, hipMemcpyDeviceToHost));
		CUDA_CALL(hipMemcpy(sumRed, dev_sumRed, PALETTE_BYTES, hipMemcpyDeviceToHost));
		CUDA_CALL(hipMemcpy(sumGreen, dev_sumGreen, PALETTE_BYTES, hipMemcpyDeviceToHost));
		CUDA_CALL(hipMemcpy(sumBlue, dev_sumBlue, PALETTE_BYTES, hipMemcpyDeviceToHost));
		CUDA_CALL(hipMemcpy(pixelClusterCounter, dev_pixelClusterCounter, PALETTE_BYTES, hipMemcpyDeviceToHost));

		//printf("Kmeans code ran in: %f msecs.\n", timer.Elapsed());
		printf("\n");

		printf("Pixels por centroide:\n");
		for(int k = 0; k < nCentroids; k++){
			printf("%d) centroid: %d pixels\n", k, pixelClusterCounter[k]);
		}
		printf("\n");
		printf("Centroides Finales:\n");
		for(int i = 0; i < nCentroids; i++) {
			printf("%d) [ %d, %d, %d ]\n",i, redCentroid[i], greenCentroid[i], blueCentroid[i]);
		}
		//Imprimir imagen resultado
		imprimir_resultado(img_name,labelArray, redCentroid, greenCentroid,  blueCentroid,  size);
		//guardar imagen

		free(red);
		free(green);
		free(blue);
		free(redCentroid);
		free(greenCentroid);
		free(blueCentroid);
		free(labelArray);
		free(sumRed);
		free(sumGreen);
		free(sumBlue);
		free(pixelClusterCounter);

		CUDA_CALL(hipFree(dev_Red));
		CUDA_CALL(hipFree(dev_Green));
		CUDA_CALL(hipFree(dev_Blue));
		CUDA_CALL(hipFree(dev_tempRedCentroid));
		CUDA_CALL(hipFree(dev_tempGreenCentroid));
		CUDA_CALL(hipFree(dev_tempBlueCentroid));
		CUDA_CALL(hipFree(dev_labelArray));
		CUDA_CALL(hipFree(dev_sumRed));
		CUDA_CALL(hipFree(dev_sumGreen));
		CUDA_CALL(hipFree(dev_sumBlue));
		CUDA_CALL(hipFree(dev_pixelClusterCounter));
	//_______________________________________
	double fps = 60;
    int delay = 1000 / fps;
    while (true){
        if(waitKey(delay) == 27) break;
    }

	return 0;
}


int main(){

	string img_name="img.jpg";
	//imgname.centorides,iteraciones
	unsigned t0, t1;
	t0=clock();
	run(img_name,5,40);//2,5,10
	t1 = clock();
	double time = (double(t1-t0)/CLOCKS_PER_SEC);
	cout << "Tiempo de ejecucion: " << time << endl;
	return 0;
}