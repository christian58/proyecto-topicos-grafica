#include "hip/hip_runtime.h"
//nvcc mat.cu  -o m `pkg-config opencv --cflags --libs`; ./m
#include <opencv2/highgui/highgui.hpp>
#include <bits/stdc++.h>
using namespace cv;
using namespace std;
#define THREADS_PER_BLOCK 1024//1024


//=======================CUDA================================
__global__ void k_gris(int *a, int *b, int *c, int value, int tam) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if( index < tam){
		int t = (a[index] + b[index] + c[index])/3;
		a[index] = t; 
		b[index] = t;
		c[index] = t;
	}
}

__global__ void k_contaste(int *a, int *b, int *c, float value, int tam) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if( index < tam){
        a[index] *= value; 
        b[index] *= value;
        c[index] *= value;
    }
}

__global__ void k_brillo(int *a, int *b, int *c, int value, int tam) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if( index < tam){
		a[index] += value; 
		b[index] += value;
		c[index] += value;
	}
}

void CUDA_process_img(int *A, int *B, int* C,int value, int rows, int cols){
	int *d_A, *d_B, *d_C;
	int nElem = rows * cols;
	int size = nElem * sizeof(int);
	//Allocate device memory for matrices
	hipMalloc((void **) &d_A, size);
	hipMalloc((void **) &d_B, size);
	hipMalloc((void **) &d_C, size);
	//Copy B and C to device memory
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
	hipMemcpy(d_C, C, size, hipMemcpyHostToDevice);
	//run
	k_gris<<<(nElem+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_A, d_B, d_C,value, nElem);//run
    k_contaste<<<(nElem+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_A, d_B, d_C,1.9, nElem);//run
	k_brillo<<<(nElem+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_A, d_B, d_C,-100, nElem);//run
	
	hipMemcpy(A, d_A, size, hipMemcpyDeviceToHost);
	hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);
	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	//Free device matrices
	hipFree(d_B);
	hipFree(d_A);
	hipFree(d_C);
}

//======================OPEN-CV=================================
//brillo en serial
void CPU_add_brillo(int *R,int *G,int *B, int value, int rows, int cols){
    for (int i=0; i<rows; i++){
        for(int j=0; j<cols; j++){
            R[cols * i + j] += value;
            G[cols * i + j] += value;
            B[cols * i + j] += value;
        }
    }
}

Mat brillo_cuda(Mat &image, int value){
    int rows = image.rows;
    int cols = image.cols;
    int nElem = rows * cols;
    int * R = (int *) malloc(nElem * sizeof(int));
    int * G = (int *) malloc(nElem * sizeof(int));
    int * B = (int *) malloc(nElem * sizeof(int));

    //load IMG
    for( int i = 0; i < image.rows; i++ ){
        for( int j = 0; j < image.cols; j++ ){
        	int r = image.at<Vec3b>(i,j)[0];
        	int g = image.at<Vec3b>(i,j)[1];
        	int b = image.at<Vec3b>(i,j)[2];
            R[cols*i+j] = r;
            G[cols*i+j] = g;
            B[cols*i+j] = b;

        }
    }
    CUDA_process_img(R,G,B,value,rows,cols);
    //CPU_add_brillo(R,G,B,value,rows,cols);

    Mat new_image = Mat::zeros( image.size(), image.type() );
    for( int i = 0; i < image.rows; i++ ){
        for( int j = 0; j < image.cols; j++ ){
            new_image.at<Vec3b>(i,j)[0] = saturate_cast<uchar>( R[cols*i+j] );
            new_image.at<Vec3b>(i,j)[1] = saturate_cast<uchar>( G[cols*i+j] );
            new_image.at<Vec3b>(i,j)[2] = saturate_cast<uchar>( B[cols*i+j] );
        }
    }
    free(R);free(G);free(B);
    return new_image;
}

void aumentar_brillo(){
    Mat img = imread("img_24.jpg", CV_LOAD_IMAGE_COLOR);
    imshow( "original", img ); 
    
    Mat img2 =brillo_cuda(img,50);
    imshow( "brillo", img2 ); 
    //waitKey(0);

    
    double fps = 60;
    // calcular el tiempo de espera entre cada imagen a mostrar
    int delay = 1000 / fps;
    while (true){
        if(waitKey(delay) == 27) break;
    }

}


int main(){
	aumentar_brillo();
	//run();
	return 0;
}
